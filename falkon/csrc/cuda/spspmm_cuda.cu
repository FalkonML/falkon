#include "spspmm_cuda.cuh"
#include "utils.cuh"

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <hipsparse.h>

#define CHECK_INPUT(x) AT_ASSERTM(x, "Input mismatch")
#define DISPATCH_SPSPMM_TYPES(TYPE, ...)                                       \
  [&] {                                                                        \
    switch (TYPE) {                                                            \
    case torch::ScalarType::Float: {                                           \
      using scalar_t = float;                                                  \
      const auto &cusparseXcsrgemm2_bufferSizeExt =                            \
          hipsparseScsrgemm2_bufferSizeExt;                                     \
      const auto &cusparseXcsrgemm2 = hipsparseScsrgemm2;                       \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case torch::ScalarType::Double: {                                          \
      using scalar_t = double;                                                 \
      const auto &cusparseXcsrgemm2_bufferSizeExt =                            \
          hipsparseDcsrgemm2_bufferSizeExt;                                     \
      const auto &cusparseXcsrgemm2 = hipsparseDcsrgemm2;                       \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    default:                                                                   \
      AT_ERROR("Not implemented for '", toString(TYPE), "'");                  \
    }                                                                          \
  }()


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
spspmm_cuda(torch::Tensor rowptrA, torch::Tensor colA, torch::Tensor valA,
            torch::Tensor rowptrB, torch::Tensor colB, torch::Tensor valB, 
	        int64_t K) {
  /* Input checks: all matrices should be in CSR format, matrix `D` is not used.
   * C = alpha*A*B + beta*D
   * A: m x k
   * B: k x n
   * D: m x n
   * C: m x n
   */
  CHECK_CUDA(rowptrA);
  CHECK_CUDA(colA);
  CHECK_CUDA(valA);
  CHECK_CUDA(rowptrB);
  CHECK_CUDA(colB);
  CHECK_CUDA(valB);

  CHECK_INPUT(rowptrA.dim() == 1);
  CHECK_INPUT(colA.dim() == 1);
  CHECK_INPUT(valA.dim() == 1);
  CHECK_INPUT(valA.size(0) == colA.size(0));

  CHECK_INPUT(rowptrB.dim() == 1);
  CHECK_INPUT(colB.dim() == 1);
  CHECK_INPUT(valB.dim() == 1);
  CHECK_INPUT(valB.size(0) == colB.size(0));

  auto scalar_type = valA.scalar_type();
  /*
  * Summary of the necessary steps
  * 1. Allocate buffer for working-memory of size given by the cusparseXcsrgemm2_bufferSizeExt function
  * 2. Compute the row-pointers of the output C with function hipsparseXcsrgemm2Nnz. This calculates the nnzC
  * 4. allocates csrValC and csrColIndC of nnzC elements respectively, and fill them with the
  *    cusparseXcsrgemm2 function
  */
  auto handle = at::cuda::getCurrentCUDASparseHandle();
  hipsparseStatus_t status;
  hipError_t cuda_status;
  at::DeviceGuard g(rowptrA.get_device());

  // Creates default matrix descriptor (0-based and GENERAL matrix)
  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);

  // Convert indices to int (could be long at input)
  rowptrA = rowptrA.toType(torch::kInt);
  colA = colA.toType(torch::kInt);
  rowptrB = rowptrB.toType(torch::kInt);
  colB = colB.toType(torch::kInt);

  int64_t M = rowptrA.numel() - 1;
  int64_t N = rowptrB.numel() - 1;
  auto rowptrA_data = rowptrA.data_ptr<int>();
  auto colA_data = colA.data_ptr<int>();
  auto rowptrB_data = rowptrB.data_ptr<int>();
  auto colB_data = colB.data_ptr<int>();
  int nnzA = colA.numel();
  int nnzB = colB.numel();
  // These values will be computed by this function
  torch::Tensor rowptrC, colC, valC;
  int nnzC;
  int *nnzTotalDevHostPtr = &nnzC;

  // Step 1: Create an opaque structure.
  csrgemm2Info_t info = NULL;
  hipsparseCreateCsrgemm2Info(&info);

  // Step 2: Allocate buffer for `csrgemm2Nnz` and `csrgemm2`.
  size_t bufferSize;
  DISPATCH_SPSPMM_TYPES(scalar_type, [&] {
    scalar_t alpha = (scalar_t)1.0;
    status = cusparseXcsrgemm2_bufferSizeExt(handle, M, N, K, &alpha,
        descr, colA.numel(), rowptrA_data, colA_data,
        descr, colB.numel(), rowptrB_data, colB_data,
        NULL, descr,  0, NULL, NULL, // Describes matrix D which is not used
        info, &bufferSize);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      hipsparseDestroyMatDescr(descr);
      hipsparseDestroyCsrgemm2Info(info);
      AT_ERROR("cusparse csrgemm2_bufferSizeExt function failed with error code '", status, "'.");
    }

    void *buffer = NULL;
    cuda_status = hipMalloc(&buffer, bufferSize);
    if (cuda_status != hipSuccess) {
      hipsparseDestroyMatDescr(descr);
      hipsparseDestroyCsrgemm2Info(info);
      AT_ERROR("cuda malloc failed with error code '", cuda_status, "'.");
    }

    // Step 3: Compute CSR row pointer. This will fill `rowptrC_data` and `nnzC`
    rowptrC = torch::empty(M + 1, rowptrA.options());
    auto rowptrC_data = rowptrC.data_ptr<int>();
    status = hipsparseXcsrgemm2Nnz(handle, M, N, K,
        descr, colA.numel(), rowptrA_data, colA_data,
        descr, colB.numel(), rowptrB_data, colB_data,
        descr, 0, NULL, NULL,
        descr, rowptrC_data, nnzTotalDevHostPtr,
        info, buffer);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      hipFree(buffer);
      hipsparseDestroyMatDescr(descr);
      hipsparseDestroyCsrgemm2Info(info);
      AT_ERROR("cusparse csrgemm2Nnz function failed with error code '", status, "'.");
    }

    // Step 4: Compute CSR entries.
    colC = torch::empty(nnzC, rowptrC.options());
    auto colC_data = colC.data_ptr<int>();

    valC = torch::empty(nnzC, valA.options());
    auto valC_data = valC.data_ptr<scalar_t>();
    auto valA_data = valA.data_ptr<scalar_t>();
    auto valB_data = valB.data_ptr<scalar_t>();

    status = cusparseXcsrgemm2(handle, M, N, K, &alpha,
        descr, colA.numel(), valA_data, rowptrA_data, colA_data,
        descr, colB.numel(), valB_data, rowptrB_data, colB_data,
        NULL, descr, 0, NULL, NULL, NULL,  // Describes matrix D
        descr, valC_data, rowptrC_data, colC_data,
        info, buffer);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      hipFree(buffer);
      hipsparseDestroyMatDescr(descr);
      hipsparseDestroyCsrgemm2Info(info);
      AT_ERROR("cusparse csrgemm2 function failed with error code '", status, "'.");
    }
    hipFree(buffer);
  });

  // Step 5: Free the opaque structure.
  hipsparseDestroyCsrgemm2Info(info);
  hipsparseDestroyMatDescr(descr);

  return std::make_tuple(rowptrC, colC, valC);
}
