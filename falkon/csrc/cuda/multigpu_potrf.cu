#include "hip/hip_runtime.h"
#include "multigpu_potrf.h"

#include <thread>
#include <atomic>
#include <algorithm>
#include <vector>
#include <set>
#include <stdio.h>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "utils.cuh"

//#define DEBUG 1

#ifndef TORCH_CUSOLVER_CHECK
#define TORCH_CUSOLVER_CHECK(EXPR)                              \
  do {                                                          \
    hipsolverStatus_t __err = EXPR;                              \
    TORCH_CHECK(__err == HIPSOLVER_STATUS_SUCCESS,               \
                "CUDA error: ",                                 \
                cusolverGetErrorString(__err),                  \
                " when calling `" #EXPR "`");                   \
  } while (0)

const char* cusolverGetErrorString(hipsolverStatus_t error) {
  if (error == HIPSOLVER_STATUS_SUCCESS) {
    return "HIPBLAS_STATUS_SUCCESS";
  }
  if (error == HIPSOLVER_STATUS_NOT_INITIALIZED) {
    return "HIPSOLVER_STATUS_NOT_INITIALIZED";
  }
  if (error == HIPSOLVER_STATUS_ALLOC_FAILED) {
    return "HIPSOLVER_STATUS_ALLOC_FAILED";
  }
  if (error == HIPSOLVER_STATUS_INVALID_VALUE) {
    return "HIPSOLVER_STATUS_INVALID_VALUE";
  }
  if (error == HIPSOLVER_STATUS_ARCH_MISMATCH) {
    return "HIPSOLVER_STATUS_ARCH_MISMATCH";
  }
  if (error == HIPSOLVER_STATUS_EXECUTION_FAILED) {
    return "HIPSOLVER_STATUS_EXECUTION_FAILED";
  }
  if (error == HIPSOLVER_STATUS_INTERNAL_ERROR) {
    return "HIPSOLVER_STATUS_INTERNAL_ERROR";
  }
  if (error == HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED) {
    return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
  }
  return "<unknown>";
}
#endif


#define TORCH_CUDABLAS_CHECK(EXPR)                                      \
  do {                                                          \
    hipblasStatus_t __err = EXPR;                              \
    TORCH_CHECK(__err == HIPBLAS_STATUS_SUCCESS,                 \
                "CuBLAS error: ",                               \
                cublasGetErrorString(__err),                    \
                " when calling `" #EXPR "`");                   \
  } while (0)


const char* cublasGetErrorString(hipblasStatus_t error) {
  if (error == HIPBLAS_STATUS_SUCCESS) {
    return "HIPBLAS_STATUS_SUCCESS";
  }
  if (error == HIPBLAS_STATUS_NOT_INITIALIZED) {
    return "HIPBLAS_STATUS_NOT_INITIALIZED";
  }
  if (error == HIPBLAS_STATUS_ALLOC_FAILED) {
    return "HIPBLAS_STATUS_ALLOC_FAILED";
  }
  if (error == HIPBLAS_STATUS_INVALID_VALUE) {
    return "HIPBLAS_STATUS_INVALID_VALUE";
  }
  if (error == HIPBLAS_STATUS_ARCH_MISMATCH) {
    return "HIPBLAS_STATUS_ARCH_MISMATCH";
  }
  if (error == HIPBLAS_STATUS_MAPPING_ERROR) {
    return "HIPBLAS_STATUS_MAPPING_ERROR";
  }
  if (error == HIPBLAS_STATUS_EXECUTION_FAILED) {
    return "HIPBLAS_STATUS_EXECUTION_FAILED";
  }
  if (error == HIPBLAS_STATUS_INTERNAL_ERROR) {
    return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }
  if (error == HIPBLAS_STATUS_NOT_SUPPORTED) {
    return "HIPBLAS_STATUS_NOT_SUPPORTED";
  }
#ifdef HIPBLAS_STATUS_UNKNOWN
  if (error == HIPBLAS_STATUS_UNKNOWN) {
    return "HIPBLAS_STATUS_UNKNOWN";
  }
#endif
  return "<unknown>";
}


/* CUDA CallBacks */
struct callBackData {
    std::atomic<int> *work_unit;
    const int x;
    const int y;
    const int callee;
};

void CUDART_CB copyCallBack(hipStream_t stream, hipError_t error, void *data) {
    callBackData *tmp = (callBackData *)(data);
#ifdef DEBUG
    fprintf(stderr, "Incrementing work unit at [%d, %d] callee: %d - from %d\n", tmp->x, tmp->y, tmp->callee, tmp->work_unit->load());
#endif
    std::atomic_fetch_add(tmp->work_unit, 1);
}

/* cu* library data and functions */
static constexpr double const oned = 1.0;
static constexpr double const moned = -1.0;
static constexpr float const onef = 1.0;
static constexpr float const monef = -1.0;

/* POTRF Buffer Size */
template<typename scalar_t>
inline int potrf_buffer_size(const hipsolverHandle_t cusolver_handle, const int mbs)
{ throw std::invalid_argument("scalar_t"); }
template<>
inline int potrf_buffer_size<double>(const hipsolverHandle_t cusolver_handle, const int mbs) {
    int potrf_buf_size;
    TORCH_CUSOLVER_CHECK(hipsolverDnDpotrf_bufferSize(
        /*handle=*/cusolver_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*n=*/mbs,
        /*A=*/NULL,
        /*lda=*/mbs,
        /*Lwork=*/&potrf_buf_size
    ));
    return potrf_buf_size;
}
template<>
inline int potrf_buffer_size<float>(const hipsolverHandle_t cusolver_handle, const int mbs) {
    int potrf_buf_size;
    TORCH_CUSOLVER_CHECK(hipsolverDnSpotrf_bufferSize(
        /*handle=*/cusolver_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*n=*/mbs,
        /*A=*/NULL,
        /*lda=*/mbs,
        /*Lwork=*/&potrf_buf_size
    ));
    return potrf_buf_size;
}


/* POTRF */
template<typename scalar_t>
inline void potrf(const hipsolverHandle_t cusolver_handle, const int mbs,
                  const blockAlloc &block_alloc, scalar_t *block_ptr, scalar_t *workspace,
                  const int workspace_size, int *potrf_info, int &potrf_info_h, hipStream_t stream)
{ throw std::invalid_argument("scalar_t"); }
template<>
inline void potrf<double>(
           const hipsolverHandle_t cusolver_handle,
           const int mbs,
           const blockAlloc &block_alloc,
           double *block_ptr,
           double *workspace,
           const int workspace_size,
           int *potrf_info,
           int &potrf_info_h,
           hipStream_t stream)
{
    TORCH_CUSOLVER_CHECK(hipsolverDnDpotrf(
        /*handle=*/cusolver_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*n=*/block_alloc.size,
        /*A=*/block_ptr,
        /*lda=*/mbs,
        /*workspace=*/workspace,
        /*Lwork=*/workspace_size,
        /*devInfo=*/potrf_info
    ));
    //C10_CUDA_CHECK(hipMemcpyAsync(&potrf_info_h, potrf_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    potrf_info_h = 0;
}
template<>
inline void potrf<float>(
           const hipsolverHandle_t cusolver_handle,
           const int mbs,
           const blockAlloc &block_alloc,
           float *block_ptr,
           float *workspace,
           const int workspace_size,
           int *potrf_info,
           int &potrf_info_h,
           hipStream_t stream)
{
    TORCH_CUSOLVER_CHECK(hipsolverDnSpotrf(
        /*handle=*/cusolver_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*n=*/block_alloc.size,
        /*A=*/block_ptr,
        /*lda=*/mbs,
        /*workspace=*/workspace,
        /*Lwork=*/workspace_size,
        /*devInfo=*/potrf_info
    ));
    //C10_CUDA_CHECK(hipMemcpyAsync(&potrf_info_h, potrf_info, sizeof(int), hipMemcpyDeviceToHost, stream));
    potrf_info_h = 0;
}


/* TRSM (cuBLAS) */
template<typename scalar_t>
inline void trsm(const hipblasHandle_t cublas_handle,
          const blockAlloc &i_alloc, const blockAlloc &b_alloc,
          scalar_t* i_block, scalar_t* b_block, const int mbs)
{ throw std::invalid_argument("scalar_t"); }
template<>
inline void trsm<double>(
          const hipblasHandle_t cublas_handle,
          const blockAlloc &i_alloc,
          const blockAlloc &b_alloc,
          double* i_block,
          double* b_block,
          const int mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasDtrsm(
        /*handle=*/cublas_handle, /*side=*/HIPBLAS_SIDE_RIGHT, /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*trans=*/HIPBLAS_OP_T, /*diag=*/HIPBLAS_DIAG_NON_UNIT,
        /*m=*/b_alloc.size, /*n=*/i_alloc.size, /*alpha=*/&oned,
        /*A=*/i_block, /*lda=*/mbs, /*B=*/b_block, /*ldb=*/mbs
    ));
}
template<>
inline void trsm<float>(
          const hipblasHandle_t cublas_handle,
          const blockAlloc &i_alloc,
          const blockAlloc &b_alloc,
          float* i_block,
          float* b_block,
          const int mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasStrsm(
        /*handle=*/cublas_handle, /*side=*/HIPBLAS_SIDE_RIGHT, /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*trans=*/HIPBLAS_OP_T, /*diag=*/HIPBLAS_DIAG_NON_UNIT,
        /*m=*/b_alloc.size, /*n=*/i_alloc.size, /*alpha=*/&onef,
        /*A=*/i_block, /*lda=*/mbs, /*B=*/b_block, /*ldb=*/mbs
    ));
}


/* GEMM (cuBLAS) */
template<typename scalar_t>
inline void gemm(const hipblasHandle_t cublas_handle,
                 const blockAlloc &b_alloc, const blockAlloc &y_alloc, const blockAlloc &i_alloc,
                 scalar_t* b_block, scalar_t* y_block, scalar_t* out_buf, const int mbs)
{ throw std::invalid_argument("scalar_t"); }
template<>
inline void gemm<double>(
        const hipblasHandle_t cublas_handle,
        const blockAlloc &b_alloc,
        const blockAlloc &y_alloc,
        const blockAlloc &i_alloc,
        double*           b_block,
        double*           y_block,
        double*           out_buf,
        const int         mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasDgemm(
        /*handle=*/cublas_handle,
        /*transa=*/HIPBLAS_OP_N,
        /*transb=*/HIPBLAS_OP_T,
        /*m=*/b_alloc.size,
        /*n=*/y_alloc.size,
        /*k=*/i_alloc.size,
        /*alpha=*/&moned,
        /*A=*/b_block,
        /*lda=*/mbs,
        /*B=*/y_block,
        /*ldb=*/mbs,
        /*beta=*/&oned,
        /*C=*/out_buf,
        /*ldc=*/mbs
    ));
}
template<>
inline void gemm<float>(
        const hipblasHandle_t cublas_handle,
        const blockAlloc &b_alloc,
        const blockAlloc &y_alloc,
        const blockAlloc &i_alloc,
        float*            b_block,
        float*            y_block,
        float*            out_buf,
        const int         mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasSgemm(
        /*handle=*/cublas_handle,
        /*transa=*/HIPBLAS_OP_N,
        /*transb=*/HIPBLAS_OP_T,
        /*m=*/b_alloc.size,
        /*n=*/y_alloc.size,
        /*k=*/i_alloc.size,
        /*alpha=*/&monef,
        /*A=*/b_block,
        /*lda=*/mbs,
        /*B=*/y_block,
        /*ldb=*/mbs,
        /*beta=*/&onef,
        /*C=*/out_buf,
        /*ldc=*/mbs
    ));
}


/* SYRK (cuBLAS) */
template<typename scalar_t>
inline void syrk(const hipblasHandle_t cublas_handle,
                 const blockAlloc &i_alloc, const blockAlloc &b_alloc,
                 scalar_t* b_block, scalar_t* out_buf, const int mbs)
{ throw std::invalid_argument("scalar_t"); }
template<>
inline void syrk<double>(
        const hipblasHandle_t cublas_handle,
        const blockAlloc &i_alloc,
        const blockAlloc &b_alloc,
        double*           b_block,
        double*           out_buf,
        const int         mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasDsyrk(
        /*handle=*/cublas_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*trans=*/HIPBLAS_OP_N,
        /*n=*/b_alloc.size,
        /*k=*/i_alloc.size,
        /*alpha=*/&moned,
        /*A=*/b_block,
        /*lda=*/mbs,
        /*beta=*/&oned,
        /*C=*/out_buf,
        /*ldc=*/mbs
    ));
}
template<>
inline void syrk<float>(
        const hipblasHandle_t cublas_handle,
        const blockAlloc &i_alloc,
        const blockAlloc &b_alloc,
        float*            b_block,
        float*            out_buf,
        const int         mbs)
{
    TORCH_CUDABLAS_CHECK(hipblasSsyrk(
        /*handle=*/cublas_handle,
        /*uplo=*/HIPBLAS_FILL_MODE_LOWER,
        /*trans=*/HIPBLAS_OP_N,
        /*n=*/b_alloc.size,
        /*k=*/i_alloc.size,
        /*alpha=*/&monef,
        /*A=*/b_block,
        /*lda=*/mbs,
        /*beta=*/&onef,
        /*C=*/out_buf,
        /*ldc=*/mbs
    ));
}


/* Data-loading helper functions */
template <typename scalar_t>
static inline void load_block(
        torch::Tensor &data_h,
        scalar_t*     &data_d,
        const blockAlloc& alloc_i,
        const blockAlloc& alloc_j,
        const int mbs,
        const hipStream_t stream)
{
    const int64_t si = data_h.stride(0);
    const int64_t sj = data_h.stride(1);
    scalar_t *data_h_ptr = data_h.data_ptr<scalar_t>();
    const uint64_t offset = si * alloc_i.start + sj * alloc_j.start;
    TORCH_CUDABLAS_CHECK(hipblasSetMatrixAsync(
        /*rows=*/alloc_i.size,
        /*cols=*/alloc_j.size,
        /*elem_size=*/sizeof(scalar_t),
        /*A=*/(void *)(data_h_ptr + offset),
        /*lda=*/sj,
        /*B=*/(void *)data_d,
        /*ldb=*/mbs,
        /*stream=*/stream
    ));
}

template <typename scalar_t>
static inline void get_block(
        scalar_t*     &data_d,
        torch::Tensor &data_h,
        const blockAlloc& alloc_i,
        const blockAlloc& alloc_j,
        const int mbs,
        const hipStream_t stream)
{
    const int64_t si = data_h.stride(0);
    const int64_t sj = data_h.stride(1);
    scalar_t *data_h_ptr = data_h.data_ptr<scalar_t>();
    const uint64_t offset = si * alloc_i.start + sj * alloc_j.start;
    TORCH_CUDABLAS_CHECK(hipblasGetMatrixAsync(
        /*rows=*/alloc_i.size,
        /*cols=*/alloc_j.size,
        /*elem_size=*/sizeof(scalar_t),
        /*A=*/(void *)data_d,
        /*lda=*/mbs,
        /*B=*/(void *)(data_h_ptr + offset),
        /*ldb=*/sj,
        /*stream=*/stream
    ));
}

template <typename scalar_t>
static inline void opt_load_block(
        torch::Tensor &data_h,
        scalar_t*     &data_d,
        const int block_id,
        std::set<int> &col0_fill,
        const blockAlloc& alloc_i,
        const blockAlloc& alloc_j,
        const int mbs,
        const hipStream_t stream)
{
    if (col0_fill.find(block_id) == col0_fill.end()) {
        load_block<scalar_t>(data_h, data_d, alloc_i, alloc_j, mbs, stream);
        col0_fill.insert(block_id);
    }
}


/* Main parallel POTRF function */
void parallel_potrf_runner(int device_id,
                           std::vector<std::vector<std::atomic<int>>> &work,
                           torch::Tensor &A,
                           std::vector<blockAlloc> &allocs,
                           hipsolverHandle_t cusolver_handle)
{
    // CUDA devices and stream
    c10::cuda::CUDAGuard g(device_id);
    at::cuda::CUDAStream s1 = at::cuda::getStreamFromPool(false, device_id);
    hipStream_t s1_c = s1.stream();
    at::cuda::CUDAStream s2 = at::cuda::getStreamFromPool(false, device_id);
    hipStream_t s2_c = s2.stream();
    at::cuda::CUDAStream s3 = at::cuda::getStreamFromPool(false, device_id);
    hipStream_t s3_c = s3.stream();
    at::cuda::CUDAStreamGuard g0(s1);

    // Fetch cuBLAS handle and set cuBLAS, cuSOLVER streams to s1
    const auto cublas_handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t orig_cublas_stream;
    TORCH_CUDABLAS_CHECK(hipblasGetStream(cublas_handle, &orig_cublas_stream));
    TORCH_CUDABLAS_CHECK(hipblasSetStream(cublas_handle, s1_c));

    hipStream_t orig_cusolver_stream;
    TORCH_CUSOLVER_CHECK(hipsolverGetStream(cusolver_handle, &orig_cusolver_stream));
    TORCH_CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, s1_c));

    const auto scalar_type = A.scalar_type();
    const int k = allocs.size();

    const int mbs = (*std::max_element(allocs.begin(), allocs.end(), [] (blockAlloc lhs, blockAlloc rhs) {
        return lhs.size < rhs.size;
    })).size;
    const uint64_t mbs_sq = mbs*mbs;
    // Figure out `my_blocks` the blocks of the current stage
    std::vector<blockAlloc> my_blocks;
    std::set<int> my_block_ids;
    for (auto &block : allocs) {
        if (block.device == device_id) {
            my_blocks.push_back(block);
            my_block_ids.insert(block.id);
        }
    }
    std::map<std::pair<int, int>, callBackData> callback_data;
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < k; j++) {
            const callBackData cback_data = {.work_unit = &(work[i][j]), .x = i, .y = j, .callee = -1};
            callback_data.insert(std::pair<std::pair<int, int>, callBackData>(std::pair<int, int>(i, j), cback_data));
        }
    }

    // col0_fill keeps track of the 'current' column: which blocks are loaded or not.
    std::set<int> col0_fill;
    
    // First GPU buffer allocation.
    const uint64_t buf_size = mbs_sq * (k + k + 1);
    const auto buf_opt = torch::TensorOptions()
        .dtype(A.dtype())
        .device(torch::kCUDA, device_id)
        .layout(torch::kStrided)
        .requires_grad(false);
    const auto data_buf = torch::empty(buf_size, buf_opt);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch_parallel_potrf", [&] {
    scalar_t *A_data = A.data_ptr<scalar_t>();

    // How much workspace does potrf need:
    int potrf_buf_size = potrf_buffer_size<scalar_t>(cusolver_handle, mbs);
    const auto potrf_buf = torch::empty(potrf_buf_size, buf_opt);
    const auto potrf_info_buf = torch::zeros(1, torch::dtype(torch::kInt32).device(torch::kCUDA, device_id));

    // Data buffers
    scalar_t *data_buf_ptr = data_buf.data_ptr<scalar_t>();
    scalar_t *potrf_buf_ptr = potrf_buf.data_ptr<scalar_t>();
    int *potrf_info_buf_ptr = potrf_info_buf.data_ptr<int>();
    scalar_t *col0_h[k];
    for (int i = 0; i < k; i++) {
        col0_h[i] = data_buf_ptr;
        data_buf_ptr += mbs_sq;
    }
    scalar_t *col1_h[k];
    for (int i = 0; i < k; i++) {
        col1_h[i] = data_buf_ptr;
        data_buf_ptr += mbs_sq;
    }
    scalar_t *g_buf = data_buf_ptr;

    // Book-keeping variables (used in the loop)
    uint col_updates_left;
    uint trail_updates_left;
    int potrf_info_h;
    scalar_t **col_buf_h;
    scalar_t **next_buf_h;
    hipStream_t s_copyback;
    // Start the main loop
    for (int i = 0; i < k; i++) {
#ifdef DEBUG
        fprintf(stderr, "Starting iteration %d\n", i);
#endif
        // Setup double-buffering (via pre-inserting elements in col0_fill)
        // and number of updates.
        col_updates_left = 0;
        trail_updates_left = 0;
        for (const auto& mb : my_blocks) {
            if (mb.id > i) {
                col_updates_left += 1;
                trail_updates_left += mb.id - i;
                if (i != 0) {col0_fill.insert(mb.id);}
            }
        }
        // Switch the double-buffered col0, col1
        if (i % 2 == 0) {
            col_buf_h = col0_h;
            next_buf_h = col1_h;
            s_copyback = s2_c;
        } else {
            col_buf_h = col1_h;
            next_buf_h = col0_h;
            s_copyback = s3_c;
        }
        C10_CUDA_CHECK(hipStreamSynchronize(s_copyback));

        // 1. POTRF
        scalar_t * i_block = col_buf_h[i];
        const auto& i_alloc = allocs[i];
        if (i_alloc.device == device_id) {
            while (work[i][i] != i) { std::this_thread::yield(); }
            opt_load_block<scalar_t>(A, i_block, i, col0_fill, i_alloc, i_alloc, mbs, s1_c); // [i, i]
            potrf<scalar_t>(cusolver_handle, mbs, i_alloc, i_block, potrf_buf_ptr, potrf_buf_size,
                            potrf_info_buf_ptr, potrf_info_h, s1_c);

            C10_CUDA_CHECK(hipStreamSynchronize(s1_c));
            if (potrf_info_h != 0) {
                AT_ERROR("Cholesky decomposition failed: leading minor of order ",
                         potrf_info_h, " is not positive definite.");
            }
            get_block<scalar_t>(i_block, A, i_alloc, i_alloc, mbs, s_copyback);
            C10_CUDA_CHECK(hipStreamAddCallback(s_copyback, copyCallBack, &callback_data.at(std::pair<int, int>(i, i)), 0));
#ifdef DEBUG
            fprintf(stderr, "D:%d  Iteration %d stage %d - finished [%d, %d]\n", device_id, i, 1, i, i);
#endif
        }

        // 2. COLUMN UPDATE
        while (work[i][i] < i + 1) { std::this_thread::yield(); }
        // Keep track of which blocks we have already processed.
        // work table cannot work for this here, since it is set asynchronously.
        std::unordered_set<int> processed_idx;
        while (col_updates_left > 0) {
            for (const auto& b_alloc : my_blocks) {
                const int b = b_alloc.id;
                if (b <= i || processed_idx.find(b) != processed_idx.end() || work[b][i] != i) {
                    continue;
                }
                scalar_t *b_block = col_buf_h[b];

                opt_load_block<scalar_t>(A, i_block, i, col0_fill, i_alloc, i_alloc, mbs, s1_c); // [i, i]
                opt_load_block<scalar_t>(A, b_block, b, col0_fill, b_alloc, i_alloc, mbs, s1_c); // [b, i]
                trsm<scalar_t>(cublas_handle, i_alloc, b_alloc, i_block, b_block, mbs);

                C10_CUDA_CHECK(hipStreamSynchronize(s1_c));

                get_block<scalar_t>(b_block, A, b_alloc, i_alloc, mbs, s_copyback);
                C10_CUDA_CHECK(hipStreamAddCallback(s_copyback, copyCallBack, &callback_data.at(std::pair<int, int>(b, i)), 0));

                col_updates_left--;
                processed_idx.insert(b);
#ifdef DEBUG
                fprintf(stderr, "D:%d  Iteration %d stage %d - finished [%d, %d]\n", device_id, i, 2, b, i);
#endif
            }
        }

        // 3. TRAILING UPDATE
        // Note that this loop does not need `processed_idx` like loop 2
        // since it is processed in order. In fact the outer while loop
        // is unnecessary
#ifdef DEBUG
        fprintf(stderr, "Starting stage 3\n");
#endif
        while (trail_updates_left > 0) {
            for (const auto& b_alloc : my_blocks) {
                int b = b_alloc.id;
                if (b < i + 1) { continue; }
                while (work[b][i] != i + 1) { std::this_thread::yield(); }

                scalar_t * b_block = col_buf_h[b];
                for (int y = b; y > i; y--) {
                    while (work[y][i] != i + 1 || work[b][y] != i) { std::this_thread::yield(); }
                    const auto& y_alloc = allocs[y];
                    scalar_t *y_block = col_buf_h[y];
                    opt_load_block<scalar_t>(A, y_block, y, col0_fill, y_alloc, i_alloc, mbs, s1_c); // [y, i]
                    load_block<scalar_t>(A, g_buf, b_alloc, y_alloc, mbs, s1_c); // [b, y]
                    if (b_alloc.id != y_alloc.id) {
                        gemm<scalar_t>(cublas_handle, b_alloc, y_alloc, i_alloc, b_block, y_block, g_buf, mbs);
                    } else {
                        syrk<scalar_t>(cublas_handle, i_alloc, b_alloc, b_block, g_buf, mbs);
                    }
                    if (y == i + 1) {
                        // We are on the column which will be tackled next, can copy directly to col0
                        C10_CUDA_CHECK(
                            hipMemcpyAsync(next_buf_h[b], g_buf, mbs_sq * sizeof(scalar_t),
                            hipMemcpyDeviceToDevice, s1_c));
                        C10_CUDA_CHECK(hipStreamSynchronize(s1_c));
                        get_block<scalar_t>(next_buf_h[b], A, b_alloc, y_alloc, mbs, s_copyback);
                        C10_CUDA_CHECK(hipStreamAddCallback(s_copyback, copyCallBack, &callback_data.at(std::pair<int, int>(b, y)), 0));
                    } else {
                        // We must free the `g_buf` variable before the next round.
                        get_block<scalar_t>(g_buf, A, b_alloc, y_alloc, mbs, s1_c);
                        C10_CUDA_CHECK(hipStreamSynchronize(s1_c));
                        std::atomic_fetch_add(&work[b][y], 1);
                    }
                    trail_updates_left--;
#ifdef DEBUG
                    fprintf(stderr, "D:%d  Iteration %d stage %d - finished [%d, %d]\n", device_id, i, 3, b, y);
#endif
                }
            }
        }
        col0_fill.clear();
    }
    C10_CUDA_CHECK(hipStreamSynchronize(s1_c));
    C10_CUDA_CHECK(hipStreamSynchronize(s2_c));
    C10_CUDA_CHECK(hipStreamSynchronize(s3_c));
    });

    TORCH_CUDABLAS_CHECK(hipblasSetStream(cublas_handle, orig_cublas_stream));
    TORCH_CUSOLVER_CHECK(hipsolverSetStream(cusolver_handle, orig_cusolver_stream));
}

torch::Tensor parallel_potrf_cuda(
                  std::vector<gpuInfo> gpu_info,
                  std::vector<blockAlloc> allocations,
                  torch::Tensor &A)
{
    CHECK_CPU(A);
    // Initialize the atomic table
    int k = allocations.size();
    std::vector<std::vector<std::atomic<int>>> work(k);
    for (int i = 0; i < k; i++) {
        work[i] = std::vector<std::atomic<int>>(k);
        for (int j = 0; j < k; j++) {
            work[i][j].store(0);
        }
    }

    std::vector<std::thread> threads;
    for (const auto& gi : gpu_info) {
        threads.push_back(
                std::thread(&parallel_potrf_runner, gi.id, std::ref(work), std::ref(A), std::ref(allocations), gi.cusolver_handle));
    }

    for (auto& t : threads) {
        t.join();
    }
    return A;
}
