#include "hip/hip_runtime.h"
#include "copy_triang_cuda.h"

#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"


#define NB 64

/*
  Matrix is size * size (no support for different size than stride).
  Columns are contiguous.
  The size * size grid is subdivided into NB * size blocks (of rows).
  Each block has NB threads, so each thread copies one row into one
  column (transpose).
  Not a particularly efficient implementation!
*/
template <typename scalar_t>
__global__ void copy_simple_kernel_lower(scalar_t* __restrict__ data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[col_pos] = data[row_pos];
            col_pos++;
        }
    }
}

// Same as the _lower version, but we copy dataT to data instead!
template <typename scalar_t>
__global__ void copy_simple_kernel_upper(scalar_t* __restrict__ data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[row_pos] = data[col_pos];
            col_pos++;
        }
    }
}


torch::Tensor copy_triang_cuda(torch::Tensor &A, const bool upper) {
    CHECK_CUDA(A);
    TORCH_CHECK(A.size(0) == A.size(1), "A must be a square 2D matrix.");

    // Transpose matrix, and flip upper if matrix is C-contiguous.
    const bool fContig = is_fortran_contig(A);
    if (!fContig)
      A = torch::transpose(A, 0, 1);
    const bool bupper = fContig ? upper : !upper;
    const int64_t nx = A.size(0);
    const dim3 dimGrid(ceildiv(nx, NB));
    const dim3 dimBlock(NB);

    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "dispatch_copy_triang", [&] {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        at::DeviceGuard g(A.device());
        if (bupper) {
            copy_simple_kernel_upper<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
        } else {
            copy_simple_kernel_lower<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
        }
    });

    if (!fContig)
        A = torch::transpose(A, 0, 1);
    return A;
}
